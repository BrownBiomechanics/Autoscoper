#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math_functions.h>

#include "PSO_kernel.h"

__device__ float fitness_function(float x[])
{
	double xyzypr_manip[6] = { 0 };
	for (int i = 0; i <= NUM_OF_DIMENSIONS - 1; i++)
	{
		xyzypr_manip[i] = (double)x[i];
	} // i

	double total = PSO_FUNC(xyzypr_manip);

	//cout << "Check total function: " << total << endl;
	return (float)total;
}

__global__ void kernelUpdateParticle(float *positions, float *velocities, float *pBests, float *gBest, float r1, float r2)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i >= NUM_OF_PARTICLES * NUM_OF_DIMENSIONS)
		return;

	float rp = r1;
	float rg = r2;

	velocities[i] = OMEGA * velocities[i] + c1 * rp*(pBests[i] - positions[i]) + c2 * rg*(gBest[i%NUM_OF_DIMENSIONS] - positions[i]);
	positions[i] += velocities[i];
}

__global__ void kernelUpdatePBest(float *positions, float *pBests, float *gBest)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i >= NUM_OF_PARTICLES * NUM_OF_DIMENSIONS || i % NUM_OF_DIMENSIONS != 0)
		return;

	float tempParticle1[NUM_OF_DIMENSIONS];
	float tempParticle2[NUM_OF_DIMENSIONS];

	for (int j = 0; j < NUM_OF_DIMENSIONS; j++)
	{
		tempParticle1[j] = positions[i + j];
		tempParticle2[j] = pBests[i + j];
	}

	if (fitness_function(tempParticle1) < fitness_function(tempParticle2))
	{
		for (int j = 0; j < NUM_OF_DIMENSIONS; j++)
			pBests[i + j] = tempParticle1[j];

		if (fitness_function(tempParticle1) < fitness_function(gBest))
		{
			for (int j = 0; j < NUM_OF_DIMENSIONS; j++)
				atomicExch(gBest + j, tempParticle1[j]);
		}
	}
}

__global__ void kernelUpdateGBest(float *gBest, float *pBests)
{
	float temp[NUM_OF_DIMENSIONS];
	for (int i = 0; i < 10 * NUM_OF_DIMENSIONS; i += NUM_OF_DIMENSIONS)
	{
		for (int k = 0; k < NUM_OF_DIMENSIONS; k++)
			temp[k] = pBests[i + k];

		if (fitness_function(temp) < fitness_function(gBest))
		{
			for (int k = 0; k < NUM_OF_DIMENSIONS; k++)
				gBest[k] = temp[k];
		}
	}
}


extern "C" void cuda_pso(float *positions, float *velocities, float *pBests, float *gBest, unsigned int MAX_EPOCHS)
{
	int size = NUM_OF_PARTICLES * NUM_OF_DIMENSIONS;

	float *devPos;
	float *devVel;
	float *devPBest;
	float *devGBest;


	/*hipDeviceProp_t prop;
	int deviceNum;
	hipGetDeviceCount(&deviceNum);
	for(int i=0;i<deviceNum;i++)
	{
		hipGetDeviceProperties(&prop,i);

		if(!prop.deviceOverlap)
		{
			printf("No device will handle overlaps, so no speed up from stream.\n");
		}
	}*/

	hipMalloc((void**)&devPos, sizeof(float)*size);
	hipMalloc((void**)&devVel, sizeof(float)*size);
	hipMalloc((void**)&devPBest, sizeof(float)*size);
	hipMalloc((void**)&devGBest, sizeof(float)*NUM_OF_DIMENSIONS);

	int threadNum = 64;
	int blocksNum = NUM_OF_PARTICLES / threadNum;

	hipMemcpy(devPos, positions, sizeof(float)*size, hipMemcpyHostToDevice);
	hipMemcpy(devVel, velocities, sizeof(float)*size, hipMemcpyHostToDevice);
	hipMemcpy(devPBest, pBests, sizeof(float)*size, hipMemcpyHostToDevice);
	hipMemcpy(devGBest, gBest, sizeof(float)*NUM_OF_DIMENSIONS, hipMemcpyHostToDevice);

	//hipEvent_t start1;
	//hipEventCreate(&start1);
	//hipEvent_t stop1;
	//hipEventCreate(&stop1);
	//float msecTotal1 = 0.0f;
	for (int iter = 0; iter < MAX_EPOCHS; iter++)
	{
		kernelUpdateParticle << <blocksNum, threadNum >> > (devPos, devVel, devPBest, devGBest, getRandomClamped(), getRandomClamped());//0.000008s

		//hipEventRecord(start1, NULL);

		kernelUpdatePBest << <blocksNum, threadNum >> > (devPos, devPBest, devGBest);

		//hipEventRecord(stop1, NULL);
		//hipEventSynchronize(stop1);
		//hipEventElapsedTime(&msecTotal1, start1, stop1);
		//printf("Time elapsed:%10.10lf s\n",(double)msecTotal1/1000);
	}

	hipMemcpy(positions, devPos, sizeof(float)*size, hipMemcpyDeviceToHost);
	hipMemcpy(velocities, devVel, sizeof(float)*size, hipMemcpyDeviceToHost);
	hipMemcpy(pBests, devPBest, sizeof(float)*size, hipMemcpyDeviceToHost);
	hipMemcpy(gBest, devGBest, sizeof(float)*NUM_OF_DIMENSIONS, hipMemcpyDeviceToHost);

	hipFree(devPos);
	hipFree(devVel);
	hipFree(devPBest);
	hipFree(devGBest);
}
