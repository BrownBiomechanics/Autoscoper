#include "hip/hip_runtime.h"
// ----------------------------------
// Copyright (c) 2011, Brown University
// All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are
// met:
//
// (1) Redistributions of source code must retain the above copyright
// notice, this list of conditions and the following disclaimer.
//
// (2) Redistributions in binary form must reproduce the above copyright
// notice, this list of conditions and the following disclaimer in the
// documentation and/or other materials provided with the distribution.
//
// (3) Neither the name of Brown University nor the names of its
// contributors may be used to endorse or promote products derived from
// this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY BROWN UNIVERSITY “AS IS” WITH NO
// WARRANTIES OR REPRESENTATIONS OF ANY KIND WHATSOEVER EITHER EXPRESS OR
// IMPLIED, INCLUDING WITHOUT LIMITATION ANY WARRANTY OF DESIGN OR
// MERCHANTABILITY OR FITNESS FOR A PARTICULAR PURPOSE, EACH OF WHICH ARE
// SPECIFICALLY DISCLAIMED, NOR ANY WARRANTY OR REPRESENTATIONS THAT THE
// SOFTWARE IS ERROR FREE OR THAT THE SOFTWARE WILL NOT INFRINGE ANY
// PATENT, COPYRIGHT, TRADEMARK, OR OTHER THIRD PARTY PROPRIETARY RIGHTS.
// IN NO EVENT SHALL BROWN UNIVERSITY BE LIABLE FOR ANY DIRECT, INDIRECT,
// INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
// BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS
// OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
// ON ANY THEORY OF LIABILITY OR CAUSE OF ACTION, WHETHER IN CONTRACT,
// STRICT LIABILITY, TORT, NEGLIGENCE OR OTHERWISE, ARISING IN ANY WAY
// OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF
// SUCH DAMAGE. ANY RECIPIENT OR USER OF THIS SOFTWARE ACKNOWLEDGES THE
// FOREGOING, AND ACCEPTS ALL RISKS AND LIABILITIES THAT MAY ARISE FROM
// THEIR USE OF THE SOFTWARE.
// ---------------------------------

/// \file Ncc_kernels.cu
/// \author Andy Loomis

#include "Ncc_kernels.h"

#include <iostream>

using namespace std;

#include <cutil_inline.h>

//////// Global variables ////////

static unsigned int g_max_n = 0;

static unsigned int g_maxNumThreads = 0;

static float* d_sums = NULL;
static float* d_nums = NULL;
static float* d_den1s = NULL;
static float* d_den2s = NULL;

//////// Helper functions ////////

static void get_device_params(unsigned int n, unsigned int maxNumThreads,
                              unsigned int& numThreads,
                              unsigned int& numBlocks,
                              unsigned int& sizeMem);

static float sum(float* f, unsigned int n);

//////// Cuda kernels ////////

__global__
void sum_kernel(float* f, float* sums, unsigned int n);

__global__
void cuda_ncc_kernel(float* f, float meanF, float* g, float meanG, float* mask,
                     float* nums, float* den1s, float* den2s,
                     unsigned int n);


//////// Interface Definitions ////////

namespace xromm
{

namespace gpu
{

void ncc_init(unsigned int max_n, unsigned int maxNumThreads)
{
    if (g_max_n != max_n || g_maxNumThreads != maxNumThreads) {
        ncc_deinit();

        unsigned int numThreads, numBlocks, sizeMem;
        get_device_params(max_n, maxNumThreads, numThreads, numBlocks, sizeMem);

        cutilSafeCall(hipMalloc(&d_sums, numBlocks*sizeof(float)));
        cutilSafeCall(hipMalloc(&d_nums, max_n*sizeof(float)));
        cutilSafeCall(hipMalloc(&d_den1s, max_n*sizeof(float)));
        cutilSafeCall(hipMalloc(&d_den2s, max_n*sizeof(float)));

        g_max_n = max_n;
        g_maxNumThreads = maxNumThreads;
    }
}

void ncc_deinit()
{
    cutilSafeCall(hipFree(d_sums));
    cutilSafeCall(hipFree(d_nums));
    cutilSafeCall(hipFree(d_den1s));
    cutilSafeCall(hipFree(d_den2s));

    g_max_n = 0;
    g_maxNumThreads = 0;
}

float ncc(float* f, float* g, float* mask, unsigned int n)
{
	float nbPixel = sum(mask, n);
	float meanF = sum(f, n) / nbPixel;
	float meanG = sum(g, n) / nbPixel;

    unsigned int numThreads, numBlocks, sizeMem;
    get_device_params(n, g_maxNumThreads, numThreads, numBlocks, sizeMem);

	cuda_ncc_kernel << <numBlocks, numThreads, sizeMem >> >(f, meanF, g, meanG, mask,
                                                        d_nums, d_den1s,
                                                        d_den2s, n);

    float den = sqrt(sum(d_den1s,n)*sum(d_den2s,n));

    if (den < 1e-5) {
        return 1e5;
    }

    return sum(d_nums,n)/den;
}

} // namespace gpu

} // namespace xromm

//////// Helper Function Definitions ////////

void get_device_params(unsigned int n,
                       unsigned int maxNumThreads,
                       unsigned int& numThreads,
                       unsigned int& numBlocks,
                       unsigned int& sizeMem)
{
    numThreads = n < maxNumThreads? n: maxNumThreads;
    numBlocks = (n+numThreads-1)/numThreads;
    sizeMem = numThreads*sizeof(float);
}

float sum(float* f, unsigned int n)
{
    unsigned int numThreads, numBlocks, sizeMem;
    get_device_params(n, g_maxNumThreads, numThreads, numBlocks, sizeMem);

    while (n > 1) {
        sum_kernel<<<numBlocks, numThreads, sizeMem>>>(f, d_sums, n);
        n = numBlocks;
        get_device_params(n, g_maxNumThreads, numThreads, numBlocks, sizeMem);
        f = d_sums;
    }

    float h_sum;
    cutilSafeCall(hipMemcpy(&h_sum,
                             d_sums,
                             sizeof(float),
                             hipMemcpyDeviceToHost));
    return h_sum;
}

__global__
void sum_kernel(float* f, float* sums, unsigned int n)
{
    extern __shared__ float sdata[];

    unsigned int i = blockDim.x*blockIdx.x+threadIdx.x;

    sdata[threadIdx.x] = (i < n) ? f[i] : 0.0f;

    __syncthreads();
    for(unsigned int s = blockDim.x/2; s > 0; s >>= 1) {
        if (threadIdx.x < s) {
            sdata[threadIdx.x] += sdata[threadIdx.x + s];
        }
        __syncthreads();
    }

    if (threadIdx.x == 0) {
        sums[blockIdx.x] = sdata[0];
    }
}

__global__
void cuda_ncc_kernel(float* f, float meanF, float* g, float meanG, float* mask,
                     float* nums, float* den1s, float* den2s,
                     unsigned int n)
{
    unsigned int i = blockDim.x*blockIdx.x+threadIdx.x;

	if (i < n && mask[i] > 0.5f) {
        float fMinusMean = f[i]-meanF;
        float gMinusMean = g[i]-meanG;

        nums[i] = fMinusMean*gMinusMean;
        den1s[i] = fMinusMean*fMinusMean;
        den2s[i] = gMinusMean*gMinusMean;
    }
    else {
        nums[i] = 0.0f;
        den1s[i] = 0.0f;
        den2s[i] = 0.0f;
    }
}

